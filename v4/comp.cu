#include "hip/hip_runtime.h"
#include "comp.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

// simple CUDA error checker
#define checkCuda(err)                                                     \
  if ((err) != hipSuccess) {                                              \
    fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,          \
            hipGetErrorString(err));                                      \
    exit(-1);                                                               \
  }

// device pointers
static int *d_row_ptr, *d_col_ind;
static int *d_vis_s, *d_vis_t;
static int *d_front_s, *d_front_t;
static int *d_front_next;
static unsigned char *d_changed_s, *d_changed_t, *d_intersect;

// expand one level of BFS on whichever side
__global__ void expand_frontier_kernel(
    int N, int rank, int size,
    const int *row_ptr, const int *col_ind,
    const int *frontier_in, int *frontier_out,
    int *visited, unsigned char *changed)
{
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  // strided partition
  if (u >= N || (u % size) != rank || frontier_in[u] == 0) return;
  int start = row_ptr[u], end = row_ptr[u+1];
  for (int e = start; e < end; ++e) {
    int v = col_ind[e];
    if (visited[v] == 0) {
      if (atomicExch(&visited[v], 1) == 0) {
        frontier_out[v] = 1;
        *changed = 1;
      }
    }
  }
}

// detect any overlap in visited arrays
__global__ void check_intersect_kernel(
    int N, const int *vis1, const int *vis2, unsigned char *found)
{
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u < N && vis1[u] && vis2[u]) {
    *found = 1;
  }
}

void cudaInitGraph(int N, int M,
                   const int *h_row_ptr,
                   const int *h_col_ind)
{
  size_t rp_bytes = (N+1) * sizeof(int),
         ci_bytes = (2*M) * sizeof(int),
         nv_bytes = N * sizeof(int);

  // copy CSR
  checkCuda(hipMalloc(&d_row_ptr, rp_bytes));
  checkCuda(hipMalloc(&d_col_ind, ci_bytes));
  checkCuda(hipMemcpy(d_row_ptr, h_row_ptr, rp_bytes, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_col_ind, h_col_ind, ci_bytes, hipMemcpyHostToDevice));

  // allocate int arrays for BFS
  checkCuda(hipMalloc(&d_vis_s, nv_bytes));
  checkCuda(hipMalloc(&d_vis_t, nv_bytes));
  checkCuda(hipMalloc(&d_front_s, nv_bytes));
  checkCuda(hipMalloc(&d_front_t, nv_bytes));
  checkCuda(hipMalloc(&d_front_next, nv_bytes));

  // allocate single-byte flags
  checkCuda(hipMalloc(&d_changed_s, 1));
  checkCuda(hipMalloc(&d_changed_t, 1));
  checkCuda(hipMalloc(&d_intersect, 1));

  // zero‐initialize
  checkCuda(hipMemset(d_vis_s, 0, nv_bytes));
  checkCuda(hipMemset(d_vis_t, 0, nv_bytes));
  checkCuda(hipMemset(d_front_s, 0, nv_bytes));
  checkCuda(hipMemset(d_front_t, 0, nv_bytes));
}

void cudaInitFrontiers(int src, int dst)
{
  int one_i = 1;
  checkCuda(hipMemcpy(d_vis_s + src,   &one_i, sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_s + src, &one_i, sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_vis_t + dst,   &one_i, sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_t + dst, &one_i, sizeof(int), hipMemcpyHostToDevice));
}

void cudaExpandFrontier(int side,
                        int *h_front_in,
                        int *h_front_out,
                        int *h_vis,
                        unsigned char *h_changed,
                        int N)
{
  int *d_front   = (side == 0 ? d_front_s   : d_front_t);
  int *d_visited = (side == 0 ? d_vis_s     : d_vis_t);
  unsigned char *d_changed = (side == 0 ? d_changed_s : d_changed_t);

  // upload host frontier & clear device buffers
  checkCuda(hipMemcpy(d_front,        h_front_in,  N * sizeof(int),       hipMemcpyHostToDevice));
  checkCuda(hipMemset(d_front_next,   0,           N * sizeof(int)));
  checkCuda(hipMemset(d_changed,      0,           1));

  // launch kernel (rank=0,size=1 here; MPI driver overrides if needed)
  int threads = 256;
  int blocks  = (N + threads - 1) / threads;
  expand_frontier_kernel<<<blocks, threads>>>(N, 0, 1,
                                              d_row_ptr, d_col_ind,
                                              d_front, d_front_next,
                                              d_visited, d_changed);
  checkCuda(hipDeviceSynchronize());

  // download results
  checkCuda(hipMemcpy(h_front_out, d_front_next, N * sizeof(int),          hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_vis,       d_visited,    N * sizeof(int),          hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_changed,   d_changed,    sizeof(unsigned char),    hipMemcpyDeviceToHost));
}

void cudaCheckIntersect(int *h_vis_s,
                        int *h_vis_t,
                        unsigned char *h_found,
                        int N)
{
  checkCuda(hipMemset(d_intersect, 0, 1));
  int threads = 256, blocks = (N + threads - 1) / threads;
  check_intersect_kernel<<<blocks, threads>>>(N, d_vis_s, d_vis_t, d_intersect);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(h_found, d_intersect, 1, hipMemcpyDeviceToHost));
}

void cudaFreeGraph()
{
  hipFree(d_row_ptr);
  hipFree(d_col_ind);
  hipFree(d_vis_s);
  hipFree(d_vis_t);
  hipFree(d_front_s);
  hipFree(d_front_t);
  hipFree(d_front_next);
  hipFree(d_changed_s);
  hipFree(d_changed_t);
  hipFree(d_intersect);
}
