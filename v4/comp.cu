// comp.cu
#include <hip/hip_runtime.h>

#include <cstdio>

// Error checking macro
#define checkCuda(err) \
  if ((err) != hipSuccess) { \
    fprintf(stderr, "CUDA err %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(-1); \
  }

// Device‐global data
static int   N_glob;
static int   threads_glob, blocks_glob;
static int  *d_row_ptr, *d_col_ind;
static int  *d_vis_s, *d_vis_t, *d_front_s, *d_front_t, *d_front_next;
static int  *d_pred_s, *d_pred_t;
static unsigned char *d_changed_s, *d_changed_t, *d_intersect;
// Host pointers (set by gpuSetup)
static int *h_vis_s, *h_vis_t, *h_front_s, *h_front_t, *h_pred_s, *h_pred_t;

// Kernel: expand one frontier, record predecessors
__global__ void expand_frontier(
  int N,
  const int *row_ptr,
  const int *col_ind,
  const int *frontier_in,
        int *frontier_out,
        int *visited,
        unsigned char *changed,
        int *pred
) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u >= N || frontier_in[u] == 0) return;
  int start = row_ptr[u], end = row_ptr[u+1];
  for (int e = start; e < end; ++e) {
    int v = col_ind[e];
    int old = atomicExch(&visited[v], 1);
    if (old == 0) {
      frontier_out[v] = 1;
      *changed        = 1;
      pred[v]         = u;
    }
  }
}

// Kernel: detect intersection
__global__ void check_intersect(
  int N,
  const int *vis1,
  const int *vis2,
        unsigned char *found
) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  if (u < N && vis1[u] && vis2[u]) *found = 1;
}

extern "C" void gpuSetup(
  int N, int M,
  int *h_row_ptr, int *h_col_ind,
  int SRC, int DST,
  int threads, int blocks,
  int *hv_s, int *hv_t,
  int *hf_s, int *hf_t,
  int *hp_s, int *hp_t
) {
  N_glob       = N;
  threads_glob = threads;
  blocks_glob  = blocks;
  h_vis_s   = hv_s;
  h_vis_t   = hv_t;
  h_front_s = hf_s;
  h_front_t = hf_t;
  h_pred_s  = hp_s;
  h_pred_t  = hp_t;

  // Allocate CSR on device
  checkCuda(hipMalloc(&d_row_ptr, (N+1)*sizeof(int)));
  checkCuda(hipMalloc(&d_col_ind, M    *sizeof(int)));
  checkCuda(hipMemcpy(d_row_ptr, h_row_ptr, (N+1)*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_col_ind, h_col_ind, M    *sizeof(int), hipMemcpyHostToDevice));

  // Allocate BFS arrays
  #define ALLOC_INT(ptr) checkCuda(hipMalloc(&ptr, N*sizeof(int)))
  ALLOC_INT(d_vis_s);
  ALLOC_INT(d_vis_t);
  ALLOC_INT(d_front_s);
  ALLOC_INT(d_front_t);
  ALLOC_INT(d_front_next);
  ALLOC_INT(d_pred_s);
  ALLOC_INT(d_pred_t);
  #undef ALLOC_INT
  checkCuda(hipMalloc(&d_changed_s,  sizeof(unsigned char)));
  checkCuda(hipMalloc(&d_changed_t,  sizeof(unsigned char)));
  checkCuda(hipMalloc(&d_intersect,  sizeof(unsigned char)));

  // Initialize device arrays from host
  checkCuda(hipMemcpy(d_vis_s,   h_vis_s,   N*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_vis_t,   h_vis_t,   N*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_s, h_front_s, N*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_t, h_front_t, N*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_pred_s,  h_pred_s,  N*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_pred_t,  h_pred_t,  N*sizeof(int), hipMemcpyHostToDevice));
}

extern "C" void gpuIterate(
  unsigned char *schg,
  unsigned char *tchg,
  unsigned char *inter
) {
  unsigned char zero = 0;
  // Reset change flags
  checkCuda(hipMemcpy(d_changed_s, &zero, 1, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_changed_t, &zero, 1, hipMemcpyHostToDevice));

  // Expand source side
  checkCuda(hipMemset(d_front_next, 0, N_glob*sizeof(int)));
  expand_frontier<<<blocks_glob,threads_glob>>>(
    N_glob, d_row_ptr, d_col_ind,
    d_front_s, d_front_next,
    d_vis_s,   d_changed_s,
    d_pred_s
  );
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(d_front_s, d_front_next, N_glob*sizeof(int), hipMemcpyDeviceToDevice));

  // Expand target side
  checkCuda(hipMemset(d_front_next, 0, N_glob*sizeof(int)));
  expand_frontier<<<blocks_glob,threads_glob>>>(
    N_glob, d_row_ptr, d_col_ind,
    d_front_t, d_front_next,
    d_vis_t,   d_changed_t,
    d_pred_t
  );
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(d_front_t, d_front_next, N_glob*sizeof(int), hipMemcpyDeviceToDevice));

  // Copy device → host
  checkCuda(hipMemcpy(h_vis_s,   d_vis_s,   N_glob*sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_vis_t,   d_vis_t,   N_glob*sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_front_s, d_front_s, N_glob*sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_front_t, d_front_t, N_glob*sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_pred_s,  d_pred_s,  N_glob*sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(h_pred_t,  d_pred_t,  N_glob*sizeof(int), hipMemcpyDeviceToHost));

  // Copy change flags
  checkCuda(hipMemcpy(schg, d_changed_s, 1, hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(tchg, d_changed_t, 1, hipMemcpyDeviceToHost));

  // Detect intersection
  checkCuda(hipMemset(d_intersect, 0, 1));
  check_intersect<<<blocks_glob,threads_glob>>>(N_glob, d_vis_s, d_vis_t, d_intersect);
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipMemcpy(inter, d_intersect, 1, hipMemcpyDeviceToHost));
}

extern "C" void gpuCopyHostToDevice() {
  checkCuda(hipMemcpy(d_vis_s,   h_vis_s,   N_glob*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_vis_t,   h_vis_t,   N_glob*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_s, h_front_s, N_glob*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_front_t, h_front_t, N_glob*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_pred_s,  h_pred_s,  N_glob*sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(d_pred_t,  h_pred_t,  N_glob*sizeof(int), hipMemcpyHostToDevice));
}

extern "C" void gpuFinalize() {
  hipFree(d_row_ptr);
  hipFree(d_col_ind);
  hipFree(d_vis_s);
  hipFree(d_vis_t);
  hipFree(d_front_s);
  hipFree(d_front_t);
  hipFree(d_front_next);
  hipFree(d_pred_s);
  hipFree(d_pred_t);
  hipFree(d_changed_s);
  hipFree(d_changed_t);
  hipFree(d_intersect);
}
