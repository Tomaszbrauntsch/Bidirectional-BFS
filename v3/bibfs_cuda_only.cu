#include <cstdio>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#define checkCuda(err) \
    if((err)!=hipSuccess){ \
        fprintf(stderr,"CUDA err %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(-1); \
    }

// expand one bfs frontier using 32-bit arrays
    __global__ void expand_frontier(
    int N, const int *row_ptr, const int *col_ind,
    const int *frontier_in, int *frontier_out,
    int *visited, unsigned char *changed
    ) {
    int tid_in_block = threadIdx.x;
    int bid = blockIdx.x;
    int u = bid * blockDim.x + tid_in_block;

    // log entry for every thread
    //printf("[expand_frontier] block %2d thread %3d → global %4d\n", bid, tid_in_block, u);

    if(u >= N || frontier_in[u] == 0) {
        //printf("[expand_frontier] global %4d → skipping (out‐of‐range or not in frontier)\n", u);
        return;
    }

    //printf("[expand_frontier] Thread %4d: expanding node %4d\n", u, u);
    int start = row_ptr[u];
    int end   = row_ptr[u+1];
    for(int e = start; e < end; ++e) {
        int v = col_ind[e];
        // atomically set visited[v] to 1, old value indicates whether it was new
        int old = atomicExch(&visited[v], 1);
        if(old == 0) {
            frontier_out[v] = 1;
            *changed = 1;
            //printf("[expand_frontier] Thread %4d: discovered neighbor %4d\n", u, v);
        }
    }
}

    __global__ void check_intersect(
        int            N,
        const int     *vis1,
        const int     *vis2,
        unsigned char *found
    ) {
        int tid = threadIdx.x;
        int bid = blockIdx.x;
        int u   = bid * blockDim.x + tid;

        // log entry for every thread
        //printf("[check_intersect] block %2d thread %3d → checking node %4d\n", bid, tid, u);

        if(u < N && vis1[u] && vis2[u]){
            //printf("[check_intersect] Thread %4d: intersection FOUND at node %4d\n", u, u);
            *found = 1;
        }
}

int main(int argc, char** argv){
    if(argc < 4){
        printf("Usage: %s <bin-file> <src> <dst>\n", argv[0]);
        return 1;
    }
    const char* path = argv[1];
    int SRC = atoi(argv[2]);
    int DST = atoi(argv[3]);
    FILE* f = fopen(path, "rb");
    if(!f){ perror("fopen"); return 1; }
    unsigned int N, M;
    fread(&N, sizeof(N), 1, f);
    fread(&M, sizeof(M), 1, f);

    std::vector<std::pair<int,int>> edges;
    edges.reserve(M * 2);
    for(unsigned int i = 0; i < M; i++){
        unsigned int u, v;
        fread(&u, sizeof(u), 1, f);
        fread(&v, sizeof(v), 1, f);
        edges.emplace_back(u, v);
        edges.emplace_back(v, u);
    }
    fclose(f);

    std::vector<int> deg(N, 0);
    for(auto &e : edges) deg[e.first]++;
    std::vector<int> row_ptr(N+1, 0);
    for(int i = 0; i < N; i++) row_ptr[i+1] = row_ptr[i] + deg[i];

    std::vector<int> col_ind(edges.size());
    std::vector<int> cursor = row_ptr;
    for(auto &e : edges){
        int u = e.first, v = e.second;
        col_ind[cursor[u]++] = v;
    }

    // debug to see nieghbors
    printf("DEBUG: Node %d neighbors:", SRC);
    for(int i = row_ptr[SRC]; i < row_ptr[SRC+1]; i++) printf(" %d", col_ind[i]);
    printf("\n");
    printf("DEBUG: Node %d neighbors:", DST);
    for(int i = row_ptr[DST]; i < row_ptr[DST+1]; i++) printf(" %d", col_ind[i]);
    printf("\n");

    std::vector<char> vis_cpu(N, 0);
    std::vector<int> queue;
    queue.reserve(N);
    queue.push_back(SRC);
    vis_cpu[SRC] = 1;
    bool cpu_found = false;
    for(size_t qi = 0; qi < queue.size(); qi++){
        int u = queue[qi];
        if(u == DST){ cpu_found = true; break; }
        for(int e = row_ptr[u]; e < row_ptr[u+1]; e++){
            int v = col_ind[e];
            if(!vis_cpu[v]){
                vis_cpu[v] = 1;
                queue.push_back(v);
            }
        }
    }
    printf("DEBUG: CPU BFS found? %s, visited %zu nodes\n", cpu_found?"YES":"NO", queue.size());

    edges.clear(); deg.clear(); cursor.clear();
    int *d_row_ptr, *d_col_ind;
    checkCuda(hipMalloc(&d_row_ptr, (N+1) * sizeof(int)));
    checkCuda(hipMalloc(&d_col_ind, col_ind.size() * sizeof(int)));
    checkCuda(hipMemcpy(d_row_ptr, row_ptr.data(), (N+1) * sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_col_ind, col_ind.data(), col_ind.size() * sizeof(int), hipMemcpyHostToDevice));

    // NEED TO ALLOCATE IN 32 BITS OTHERWISE IT CRASHES
    int *d_vis_s, *d_vis_t, *d_front_s, *d_front_t, *d_front_next;
    unsigned char *d_changed_s, *d_changed_t, *d_intersect;
    checkCuda(hipMalloc(&d_vis_s,      N * sizeof(int)));
    checkCuda(hipMalloc(&d_vis_t,      N * sizeof(int)));
    checkCuda(hipMalloc(&d_front_s,    N * sizeof(int)));
    checkCuda(hipMalloc(&d_front_t,    N * sizeof(int)));
    checkCuda(hipMalloc(&d_front_next, N * sizeof(int)));
    checkCuda(hipMalloc(&d_changed_s,  sizeof(unsigned char)));
    checkCuda(hipMalloc(&d_changed_t,  sizeof(unsigned char)));
    checkCuda(hipMalloc(&d_intersect,  sizeof(unsigned char)));

    hipMemset(d_vis_s,      0, N * sizeof(int));
    hipMemset(d_vis_t,      0, N * sizeof(int));
    hipMemset(d_front_s,    0, N * sizeof(int));
    hipMemset(d_front_t,    0, N * sizeof(int));
    hipMemset(d_front_next, 0, N * sizeof(int));
    hipMemset(d_intersect,  0, sizeof(unsigned char));
    int one = 1;
    checkCuda(hipMemcpy(d_vis_s + SRC,   &one, sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_front_s + SRC, &one, sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_vis_t + DST,   &one, sizeof(int), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_front_t + DST, &one, sizeof(int), hipMemcpyHostToDevice));
    int threads = 256;
    int blocks  = (N + threads - 1) / threads;
    bool found = false;
    // std::vector<int> f_s(N), f_t(N);
    // cudaMemcpy(f_s.data(), d_front_s, N * sizeof(int), cudaMemcpyDeviceToHost);
    // cudaMemcpy(f_t.data(), d_front_t, N * sizeof(int), cudaMemcpyDeviceToHost);
    // int c_s = std::count(f_s.begin(), f_s.end(), 1);
    // int c_t = std::count(f_t.begin(), f_t.end(), 1);
    // printf("DEBUG: Initial frontier_s count = %d, frontier_t count = %d\n", c_s, c_t);

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));
    checkCuda(hipEventRecord(start));
    // alternate expanisons because only way I could make it work
    for(int iter = 0; iter < N && !found; iter++){
        unsigned char zero = 0, schg = 0, tchg = 0, inter = 0;

        hipMemcpy(d_changed_s, &zero, sizeof(zero), hipMemcpyHostToDevice);
        hipMemset(d_front_next, 0, N * sizeof(int));
        expand_frontier<<<blocks, threads>>>(N, d_row_ptr, d_col_ind, d_front_s, d_front_next, d_vis_s, d_changed_s);
        hipError_t e = hipGetLastError();
        if(e != hipSuccess)  
            printf("Kernel launch error: %s\n", hipGetErrorString(e));
        checkCuda(hipDeviceSynchronize());
        hipMemcpy(&schg, d_changed_s, sizeof(schg), hipMemcpyDeviceToHost);
        hipMemcpy(d_front_s, d_front_next, N * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(d_changed_t, &zero, sizeof(zero), hipMemcpyHostToDevice);
        hipMemset(d_front_next, 0, N * sizeof(int));
        expand_frontier<<<blocks, threads>>>(N, d_row_ptr, d_col_ind, d_front_t, d_front_next, d_vis_t, d_changed_t);
        e = hipGetLastError();
        if(e != hipSuccess)  
            printf("Kernel launch error: %s\n", hipGetErrorString(e));
        checkCuda(hipDeviceSynchronize());
        hipMemcpy(&tchg, d_changed_t, sizeof(tchg), hipMemcpyDeviceToHost);
        hipMemcpy(d_front_t, d_front_next, N * sizeof(int), hipMemcpyDeviceToDevice);
        printf("Iter %d: schg=%d, tchg=%d\n", iter, (int)schg, (int)tchg);
        hipMemcpy(d_intersect, &zero, sizeof(zero), hipMemcpyHostToDevice);
        check_intersect<<<blocks, threads>>>(N, d_vis_s, d_vis_t, d_intersect);
        checkCuda(hipDeviceSynchronize());
        hipMemcpy(&inter, d_intersect, sizeof(inter), hipMemcpyDeviceToHost);
        if(inter){ found = true; break; }

        // ggs
        if(!schg && !tchg) break;
    }
    checkCuda(hipEventRecord(stop));
    checkCuda(hipEventSynchronize(stop));
    float ms;
    checkCuda(hipEventElapsedTime(&ms, start, stop));
    printf("DEBUG: gpu bfs time = %f ms\n", ms);

    std::vector<int> vs(N), vt(N);
    hipMemcpy(vs.data(), d_vis_s, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(vt.data(), d_vis_t, N*sizeof(int), hipMemcpyDeviceToHost);
    int count_s=0, count_t=0;
    for(int i=0;i<N;i++){
      count_s += (vs[i]!=0);
      count_t += (vt[i]!=0);
    }
    printf("DEBUG: GPU visited_s count = %d\n", count_s);
    printf("DEBUG: GPU visited_t count = %d\n", count_t);
    if(vs[DST]) printf("DEBUG: source‐side actually reached DST on GPU!\n");
    if(vt[SRC]) printf("DEBUG: target‐side actually reached SRC on GPU!\n");


    printf(found ? "GPU BFS: PATH FOUND\n" : "GPU BFS: NO PATH\n");

    // cleanup
    hipFree(d_row_ptr);   hipFree(d_col_ind);
    hipFree(d_vis_s);     hipFree(d_vis_t);
    hipFree(d_front_s);   hipFree(d_front_t);
    hipFree(d_front_next);
    hipFree(d_changed_s); hipFree(d_changed_t);
    hipFree(d_intersect);
    return 0;
}
